#include "hip/hip_runtime.h"
//==============================================================================
//
//  @@-COPYRIGHT-START-@@
//
//  Copyright (c) 2016-2017, Qualcomm Innovation Center, Inc. All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software
//     without specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
//  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
//  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
//  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
//  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
//  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  @@-COPYRIGHT-END-@@
//
//==============================================================================

#include <hipblas.h>
#include <limits>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include "cuda_util.hpp"
#include "math_functions.hpp"

namespace DlQuantization
{
template <typename DTYPE>
DTYPE GetMax_gpu(const DTYPE* data, int cnt)
{
    const thrust::device_ptr<const DTYPE> ptr = thrust::device_pointer_cast(data);
    return thrust::reduce(ptr, ptr + cnt, std::numeric_limits<DTYPE>::lowest(), thrust::maximum<DTYPE>());
}

template <typename DTYPE>
DTYPE GetMin_gpu(const DTYPE* data, int cnt)
{
    const thrust::device_ptr<const DTYPE> ptr = thrust::device_pointer_cast(data);
    return thrust::reduce(ptr, ptr + cnt, std::numeric_limits<DTYPE>::max(), thrust::minimum<DTYPE>());
}

__global__ void ElementwiseMult_kernel(const float* in, size_t cnt, float factor, float* out)
{
    CUDA_KERNEL_LOOP(i, cnt)
    {
        out[i] = in[i] * factor;
    }
}

void ElementwiseMult_gpu(const float* in, size_t cnt, float factor, float* out)
{
    ElementwiseMult_kernel<<<CUDA_NUM_BLOCKS(cnt), CUDA_NUM_THREADS>>>(in, cnt, factor, out);
}

bool GemmFloat_gpu(int M, int N, int K, const float* A, const float* B, float* C, bool transposeB)
{
    hipblasHandle_t handle;
    bool success             = (HIPBLAS_STATUS_SUCCESS == hipblasCreate(&handle));
    const float alpha        = 1;
    const float beta         = 0;
    hipblasOperation_t transB = !transposeB ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    int ldb                  = !transposeB ? N : K;
    // Note that cuBLAS uses column major order, whereas C uses row major order.
    success &=
        (HIPBLAS_STATUS_SUCCESS == hipblasSgemm(handle, transB, HIPBLAS_OP_N, N, M, K, &alpha, B, ldb, A, K, &beta, C, N));
    // hipDeviceSynchronize();
    return success;
}

void* MemoryAllocation_gpu(size_t bytes)
{
    void* devPtr;
    hipMalloc(&devPtr, bytes);
    return devPtr;
}

bool MemoryFree_gpu(void* data)
{
    return hipSuccess == hipFree(data);
}

// Explicit instantiations
template double GetMax_gpu(const double* data, int cnt);

template float GetMax_gpu(const float* data, int cnt);

template double GetMin_gpu(const double* data, int cnt);

template float GetMin_gpu(const float* data, int cnt);

}   // End of namespace DlQuantization
