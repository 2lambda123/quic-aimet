
#include <hip/hip_runtime.h>
//==============================================================================
//
//  @@-COPYRIGHT-START-@@
//
//  Copyright (c) 2020, Qualcomm Innovation Center, Inc. All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software
//     without specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
//  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
//  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
//  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
//  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
//  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  @@-COPYRIGHT-END-@@
//
//==============================================================================

#ifdef GOOGLE_CUDA

#define EIGEN_USE_GPU
#define EIGEN_USE_THREADS

#include "AimetOpUtils.h"

using namespace tensorflow;

#define EIGEN_USE_GPU
typedef Eigen::GpuDevice GPUDevice;


// GPU specialization of actual computations.
template <typename T>
void copyInputTensorsToOutputTensors(const GPUDevice& d, const T* inTensor, size_t count, T* outTensor)
{
    // copy input_tensor to output_tensor
    cudaMemcpy(outTensor, inTensor, count * sizeof(float), cudaMemcpyDeviceToDevice);
}

template <typename T>
T copyLiteralToHost(const GPUDevice& d, const T* deviceValue)
{
    T hostValue;
    cudaMemcpy(&hostValue, deviceValue, sizeof(T), cudaMemcpyDeviceToHost);

    return hostValue;
}

template void copyInputTensorsToOutputTensors(const GPUDevice& d, const float* inTensor, size_t count, float* outTensor);
template int32 copyLiteralToHost<int32>(const GPUDevice&, const int32* deviceValue);
template uint64 copyLiteralToHost<uint64>(const GPUDevice&, const uint64* deviceValue);
template double copyLiteralToHost<double>(const GPUDevice&, const double* deviceValue);


#endif   // GOOGLE_CUDA